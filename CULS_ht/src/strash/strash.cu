#include "hip/hip_runtime.h"
#include "strash.h"
#include "gpu_timer.hpp"
#include "hash_table.h"

template <typename KeyT, typename ValueT>
ValueT generate_value(KeyT in) {
  return in + 1;
}

template <typename KeyT, typename ValueT, typename size_type>
__host__ void generate_uniform_unique_pairs(KeyT *keys,
                               ValueT *values,
                               size_type num_keys,
                               bool cache=false){
    std::cout << "Begin Gen: " << num_keys << std::endl;
    unsigned min_key = 0;
    unsigned seed = 1;
    // bool cache = true;
    std::string dataset_dir = "dataset";
    std::string dataset_name = std::to_string(num_keys) + "_" + std::to_string(seed);
    std::string dataset_path = dataset_dir + "/" + dataset_name;
    if (cache) {
        if (std::filesystem::exists(dataset_dir)) {
        if (std::filesystem::exists(dataset_path)) {
            std::cout << "Reading cached keys.." << std::endl;
            std::ifstream dataset(dataset_path, std::ios::binary);
            dataset.read((char*)keys, sizeof(KeyT) * num_keys);
            dataset.read((char*)values, sizeof(ValueT) * num_keys);
            dataset.close();
            return;
        }
        } else {
        std::filesystem::create_directory(dataset_dir);
        }
    }
    std::random_device rd;
    std::mt19937 rng(seed); //生成伪随机数引擎, 种子为seed
    auto max_key = std::numeric_limits<KeyT>::max() - 1; 
    std::uniform_int_distribution<KeyT> uni(min_key, max_key); //生成区间为min-max随机数字的对象
    // std::unordered_set<key_type> unique_keys;
    int count = 0;
    std::cout << "Start Random Gen" << std::endl;
    while (count < num_keys) {
        // unique_keys.insert(uni(rng)); //随机生成范围内数字插入
        // unique_keys.insert(unique_keys.size() + 1);
        keys[count++] = uni(rng);
    }
    // std::copy(unique_keys.cbegin(), unique_keys.cend(), keys);
    // std::shuffle(keys.begin(), keys.end(), rng);

    #ifdef _WIN32
    // OpenMP + windows don't allow unsigned loops
    for (uint32_t i = 0; i < unique_keys.size(); i++) {
        values[i] = generate_value<key_type, value_type>(keys[i]);
    }
    #else

    for (uint32_t i = 0; i < num_keys; i++) {
        values[i] = generate_value<KeyT,ValueT>(keys[i]);
    }
    #endif

    if (cache) {
        std::cout << "Caching.." << std::endl;
        std::ofstream dataset(dataset_path, std::ios::binary);
        dataset.write((char*)keys, sizeof(KeyT) * num_keys);
        dataset.write((char*)values, sizeof(ValueT) * num_keys);
        dataset.close();
    }
    std::cout << "Finish Random Gen" << std::endl;
}

template <typename KeyT, typename ValueT, typename size_type>
__global__ void insert_batch_kvpairs(size_type num_keys, KeyT *ht_keys, ValueT *ht_values, KeyT *keys, ValueT *values, size_type capacity){
    size_type idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < num_keys){
        insert_single_no_update(ht_keys, ht_values, keys[idx], values[idx], capacity);
    }
}

template <typename KeyT, typename ValueT, typename size_type>
__global__ void retriev_batch_kvpairs(size_type num_keys, KeyT *ht_keys, ValueT *ht_values, KeyT *keys, ValueT *values, size_type capacity){
    size_type idx = blockIdx.x * blockDim.x + threadIdx.x;
    ValueT temp;
    if(idx < num_keys){
        temp = retrieve_single(ht_keys, ht_values, keys[idx], capacity);
        if (temp != values[idx]){
            printf("Find Error!\n");
        }
    }
}

int clkAndEvaluate(uint32_t num_keys, uint32_t *keys, uint32_t *values, uint32_t capacity){
    uint32_t *g_keys;
    uint32_t *g_values;
    HashTable<uint32_t, uint32_t> ht_table(capacity);
    uint32_t* ht_keys = ht_table.get_keys_storage();
    uint32_t* ht_values = ht_table.get_values_storage();
    keys = (uint32_t*)malloc(num_keys*sizeof(uint32_t));
    values = (uint32_t*)malloc(num_keys*sizeof(uint32_t));
    generate_uniform_unique_pairs(keys, values, num_keys, true);
    // for (uint32_t i = num_keys - 100; i < num_keys; i++) std::cout<< i << ": " << keys[i] << std::endl;
    hipMalloc(&g_keys, num_keys*sizeof(uint32_t));
    hipMalloc(&g_values, num_keys*sizeof(uint32_t));
    hipMemcpy(g_keys, keys, num_keys*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(g_values, values, num_keys*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipStream_t stream;
    hipStreamCreate(&stream);
    gpu_timer timer(stream);
    timer.start_timer();
    insert_batch_kvpairs<<<NUM_BLOCKS(num_keys, THREAD_PER_BLOCK), THREAD_PER_BLOCK, 0, stream>>>(num_keys, ht_keys, ht_values, g_keys, g_values, capacity);
    // hipDeviceSynchronize();
    timer.stop_timer();
    auto insert_s = timer.get_elapsed_s();
    printf("Finished Hash, time = %lf secs\n", insert_s);
    // clock_t start = clock();
    // clock_t finish = clock();
    // printf("Finished Hash, time = %lf secs\n", (finish - start) / (double) CLOCKS_PER_SEC);
    timer.start_timer();
    retriev_batch_kvpairs<<<NUM_BLOCKS(num_keys, THREAD_PER_BLOCK), THREAD_PER_BLOCK, 0, stream>>>(num_keys, ht_keys, ht_values, g_keys, g_values, capacity);
    // hipDeviceSynchronize();
    timer.stop_timer();
    auto find_s = timer.get_elapsed_s();
    // start = clock();
    // retriev_batch_kvpairs<<<NUM_BLOCKS(num_keys, THREAD_PER_BLOCK), THREAD_PER_BLOCK>>>(num_keys, ht_keys, ht_values, g_keys, g_values, capacity);
    // hipDeviceSynchronize();
    // finish = clock();
    printf("Finished Find, time = %lf secs\n", find_s);
    return 0;
}